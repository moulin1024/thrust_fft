#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define checkCudaErrors(val)  __checkCudaErrors__ ( (val), #val, __FILE__, __LINE__ )
 
template <typename T>
inline void __checkCudaErrors__(T code, const char *func, const char *file, int line) 
{
    if (code) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, (unsigned int)code, func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

/********************************/
/* SCALE USING A CUFFT CALLBACK */
/********************************/
__device__ void scale_cufft_callback(
    void *dataOut,
    size_t offset,
    float2 element,
    void *callerInfo,
    void *sharedPtr)
{
    float2 output;

    output.x = element.x / 2;
    output.y = element.y / 2;

    ((float2*)dataOut)[offset] = output;
}

__device__
hipfftCallbackStoreC d_storeCallbackPtr = scale_cufft_callback;

int main(void){

    const int N=2;

    // --- Setting up input device vector
    thrust::device_vector<float2> d_vec(N,make_hipComplex(1.0f,2.0f));

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

    // --- Preparing the callback
    hipfftCallbackStoreC h_storeCallbackPtr;
    checkCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr,
					HIP_SYMBOL(d_storeCallbackPtr),
					sizeof(h_storeCallbackPtr)));

    // --- Associating the callback with the plan.
    hipfftResult status = hipfftXtSetCallback(plan,
				(void **)&h_storeCallbackPtr,
				HIPFFT_CB_ST_COMPLEX,
				0);
    if (status == CUFFT_LICENSE_ERROR) {
	printf("License file was not found, out of date, or invalid.\n");
	exit(EXIT_FAILURE);
    } else {
	checkCudaErrors(status);
    }

    // --- Perform in-place direct Fourier transform
    checkCudaErrors(hipfftExecC2C(plan, thrust::raw_pointer_cast(d_vec.data()),thrust::raw_pointer_cast(d_vec.data()), HIPFFT_FORWARD));
    //thrust::transform(d_vec.begin(), d_vec.end(), d_vec.begin(), scale_result((float)(2)));

    // --- Setting up output host vector
    thrust::host_vector<float2> h_vec(d_vec);

    for (int i=0; i<N; i++) printf("Element #%i: \t (%f, %f)\n",i,h_vec[i].x,h_vec[i].y);

    //Clean up
    checkCudaErrors(hipfftDestroy(plan));
}
